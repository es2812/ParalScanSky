#include "hip/hip_runtime.h"
/*
* Contar cuerpos celestes
*
* Asignatura Computación Paralela (Grado Ingeniería Informática)
* Código secuencial base
*
* @author Ana Moretón Fernández, Arturo Gonzalez-Escribano
* @author Luis Higuero Casado, Esther Cuervo Fernández
* @version v1.3
*
* (c) 2017, Grupo Trasgo, Universidad de Valladolid
*/

#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "cputils.h"

/* Substituir min por el operador */
#define min(x,y)    ((x) < (y)? (x) : (y))
#define THREADSPORBLOQUE 128

/**
* Funcion secuencial para la busqueda de mi bloque
*/
__device__ int computation(int x, int y, int columns, int* matrixData, int *matrixResult, int *matrixResultCopy){
	// Inicialmente cojo mi indice
	int result=matrixResultCopy[x*columns+y];
	if( result!= -1){
		//Si es de mi mismo grupo, entonces actualizo
		if(matrixData[(x-1)*columns+y] == matrixData[x*columns+y])
		{
			result = min (result, matrixResultCopy[(x-1)*columns+y]);
		}
		if(matrixData[(x+1)*columns+y] == matrixData[x*columns+y])
		{
			result = min (result, matrixResultCopy[(x+1)*columns+y]);
		}
		if(matrixData[x*columns+y-1] == matrixData[x*columns+y])
		{
			result = min (result, matrixResultCopy[x*columns+y-1]);
		}
		if(matrixData[x*columns+y+1] == matrixData[x*columns+y])
		{
			result = min (result, matrixResultCopy[x*columns+y+1]);
		}

		// Si el indice no ha cambiado retorna 0
		if(matrixResult[x*columns+y] == result){ return 0; }
		// Si el indice cambia, actualizo matrix de resultados con el indice adecuado y retorno 1
		else { matrixResult[x*columns+y]=result; return 1;}

	}
	return 0;
}


/*Definicion de kernels*/
__global__ void etiquetadoInicial(int *matrixResult, int *matrixResultCopy, int *matrixData, int tamMatriz){
	int indiceThread = (blockIdx.x*blockDim.x)+(threadIdx.x);

	if(indiceThread < tamMatriz){
		matrixResultCopy[indiceThread] = -1;
		matrixResult[indiceThread] = -1;

		if(matrixData[indiceThread] != 0){
			matrixResult[indiceThread] = indiceThread;
		}
	}

}

__global__ void actualizacionCopia(int *matrixResult, int *matrixResultCopy, int tamMatriz){
		int indiceThread = (blockIdx.x*blockDim.x)+(threadIdx.x);
		if(indiceThread < tamMatriz){
			if(matrixResult[indiceThread] != -1){
				matrixResultCopy[indiceThread] = matrixResult[indiceThread];
			}
		}
}

__global__ void computo(int *matrixResult, int *matrixResultCopy, int *matrixData, int filas, int columnas, int *arrayCambio){
		int indiceThread = (blockIdx.x*blockDim.x)+(threadIdx.x);
		if(indiceThread < columnas*filas){
			int x,y;
			x = indiceThread/columnas; //la fila en la que está la posicion es el entero resultante de dividir el indice por el tamaño de la fila...
			y = indiceThread%columnas; //y la columna es el resto
			arrayCambio[indiceThread] = computation(x, y, columnas, matrixData, matrixResult, matrixResultCopy);
		}
}
//Esta funcion suma todo su bloque en su posicion de auxiliar
__global__ void recuento(int *arrayCambio, int *arrayAux, int rows , int columns){
		int numbloque = blockIdx.x;
		int indiceThreadGlobal = (blockIdx.x*blockDim.x)+(threadIdx.x);

		int i;

		for (i=2;i<=blockDim.x;i=i*2){
			arrayCambio[indiceThreadGlobal]=arrayCambio[indiceThreadGlobal]+arrayCambio[indiceThreadGlobal+i/2];
			__syncthreads();
		}

		if((indiceThreadGlobal%blockDim.x)==0){
			arrayAux[numbloque]=arrayCambio[indiceThreadGlobal];
		}
}

/*Fin de kernels*/

/**
* Funcion principal
*/
int main (int argc, char* argv[])
{

	/* 1. Leer argumento y declaraciones */
	if (argc < 2) 	{
		printf("Uso: %s <imagen_a_procesar>\n", argv[0]);
		return(EXIT_SUCCESS);
	}
	char* image_filename = argv[1];

	int rows=-1;
	int columns =-1;
	int *matrixData=NULL;
	int *matrixResult=NULL;
	int *matrixResultCopy=NULL;
	int numBlocks=-1;



	/* 2. Leer Fichero de entrada e inicializar datos */

	/* 2.1 Abrir fichero */
	FILE *f = cp_abrir_fichero(image_filename);

	// Compruebo que no ha habido errores
	if (f==NULL)
	{
	   perror ("Error al abrir fichero.txt");
	   return -1;
	}

	/* 2.2 Leo valores del fichero */
	int i,j;
	fscanf (f, "%d\n", &rows);
	fscanf (f, "%d\n", &columns);
	// Añado dos filas y dos columnas mas para los bordes
	rows=rows+2;
	columns = columns+2;

	/* 2.3 Reservo la memoria necesaria para la matriz de datos */
	matrixData= (int *)malloc( rows*(columns) * sizeof(int) );
	if ( (matrixData == NULL)   ) {
 		perror ("Error reservando memoria");
	   	return -1;
	}

	/* 2.4 Inicializo matrices */
	for(i=0;i< rows; i++){
		for(j=0;j< columns; j++){
			matrixData[i*(columns)+j]=-1;
		}
	}
	/* 2.5 Relleno bordes de la matriz */
	for(i=1;i<rows-1;i++){
		matrixData[i*(columns)+0]=0;
		matrixData[i*(columns)+columns-1]=0;
	}
	for(i=1;i<columns-1;i++){
		matrixData[0*(columns)+i]=0;
		matrixData[(rows-1)*(columns)+i]=0;
	}
	/* 2.6 Relleno la matriz con los datos del fichero */
	for(i=1;i<rows-1;i++){
		for(j=1;j<columns-1;j++){
			fscanf (f, "%d\n", &matrixData[i*(columns)+j]);
		}
	}
	fclose(f);

	#ifdef WRITE
		printf("Inicializacion \n");
		for(i=0;i<rows;i++){
			for(j=0;j<columns;j++){
				printf ("%d\t", matrixData[i*(columns)+j]);
			}
			printf("\n");
		}
	#endif

	hipSetDevice(0);
	hipDeviceSynchronize();

	/* PUNTO DE INICIO MEDIDA DE TIEMPO */
	double t_ini = cp_Wtime();

//
// EL CODIGO A PARALELIZAR COMIENZA AQUI
//

	hipError_t err1, err2, err3, err4, err5; //variables para comprobación de errores.
	int tamMatriz = rows*columns;
	int numbloques=tamMatriz/THREADSPORBLOQUE + (tamMatriz%THREADSPORBLOQUE != 0);

	matrixResult= (int *)malloc( (rows)*(columns) * sizeof(int) );
	int *arrayCambio = (int *)malloc(numbloques*sizeof(int));

	if ( (matrixResult == NULL) || (arrayCambio==NULL) ) {
 		perror ("Error reservando memoria");
	   	return -1;
	}

	/*Envio de las matrices a la GPU*/
	//Inicializacion
	int *GPUmatrixResult;
	int *GPUmatrixResultCopy;
	int *GPUmatrixData;

	int *GPUArrayCambio;
	int *GPUArrayCambioAux;

	err1 = hipMalloc(&GPUmatrixResult, rows*columns*sizeof(int));
	err2 = hipMalloc(&GPUmatrixResultCopy, rows*columns*sizeof(int));
	err3 = hipMalloc(&GPUmatrixData, rows*columns*sizeof(int));

	err4 = hipMalloc(&GPUArrayCambio, rows*columns*sizeof(int));
	err5 = hipMalloc(&GPUArrayCambioAux,numbloques*sizeof(int));



	if(err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess || err4 != hipSuccess || err5!=hipSuccess){
		printf("Error en el reservado de memoria GPU\n");
		return -1;
	}

	//Envio a GPU

	err1 = hipMemcpy(GPUmatrixData, matrixData, rows*columns*sizeof(int), hipMemcpyHostToDevice);

	if(err1 != hipSuccess){
		printf("Error enviando las matrices a la GPU\n");
		return -1;
	}

	/*Definicion de grids*/


	dim3 bloque(THREADSPORBLOQUE,1);
	dim3 grid(numbloques,1);

	/* 3. Etiquetado inicial */
	etiquetadoInicial<<<grid,bloque>>>(GPUmatrixResult, GPUmatrixResultCopy, GPUmatrixData,tamMatriz);


	/* 4. Computacion */
	int t=0;
	/* 4.1 Flag para ver si ha habido cambios y si se continua la ejecucion */
	int flagCambio=1;

	/* 4.2 Busqueda de los bloques similiares */
	for(t=0; flagCambio !=0; t++){
		flagCambio=0;

		/* 4.2.1 Actualizacion copia */

		actualizacionCopia<<<grid,bloque>>>(GPUmatrixResult,GPUmatrixResultCopy,tamMatriz);

		/* 4.2.2 Computo y detecto si ha habido cambios */

		computo<<<grid,bloque>>>(GPUmatrixResult, GPUmatrixResultCopy, GPUmatrixData, rows, columns, GPUArrayCambio);

		recuento<<<grid,bloque>>>(GPUArrayCambio,GPUArrayCambioAux,rows,columns);

		//El resultado de flagCambio se guarda en un array, hacemos reduccion en el host

		err1 = hipMemcpy(arrayCambio,GPUArrayCambioAux,numbloques*sizeof(int),hipMemcpyDeviceToHost);

		if(err1 != hipSuccess){
			printf("Error copiando memoria al host %s\n",err1);
			return -1;
		}

		for(i=0;i<numbloques;i++){
		  flagCambio = arrayCambio[i];
		  if(flagCambio != 0) break;
		}

		#ifdef DEBUG
			printf("\nResultados iter %d: \n", t);
			for(i=0;i<rows;i++){
				for(j=0;j<columns;j++){
					printf ("%d\t", matrixResult[i*columns+j]);
				}
				printf("\n");
			}
		#endif
		//printf("FlagCambio%d\n",flagCambio);

	}

  //Una vez terminada la computación, se habrá generado el matrixResult final en la GPU

	err1 = hipMemcpy(matrixResult,GPUmatrixResult,rows*columns*sizeof(int),hipMemcpyDeviceToHost);

	if(err1 != hipSuccess){
		printf("Error copiando memoria al host %s\n",err1);
		return -1;
	}


	/* 4.3 Inicio cuenta del numero de bloques */
	numBlocks=0;
	for(i=1;i<rows-1;i++){
		for(j=1;j<columns-1;j++){
			if(matrixResult[i*columns+j] == i*columns+j) numBlocks++;
		}
	}

	/* Liberacion de memoria*/
	hipFree(GPUmatrixResult);
	hipFree(GPUmatrixData);
	hipFree(GPUmatrixResultCopy);
	hipFree(GPUArrayCambio);
	hipFree(GPUArrayCambioAux);

//
// EL CODIGO A PARALELIZAR TERMINA AQUI
//

	/* PUNTO DE FINAL DE MEDIDA DE TIEMPO */
	hipDeviceSynchronize();
 	double t_fin = cp_Wtime();


	/* 5. Comprobación de resultados */
  	double t_total = (double)(t_fin - t_ini);

	printf("Result: %d:%d\n", numBlocks, t);
	printf("Time: %lf\n", t_total);
	#ifdef WRITE
		printf("Resultado: \n");
		for(i=0;i<rows;i++){
			for(j=0;j<columns;j++){
				printf ("%d\t", matrixResult[i*columns+j]);
			}
			printf("\n");
		}
	#endif

	/* 6. Liberacion de memoria */
	free(matrixData);
	free(matrixResult);
	free(matrixResultCopy);

	return 0;
}
